#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hiprand/hiprand_kernel.h>
#include ""
#include "./RandomGenerator.cuh"

__global__ void setupCurand(hiprandState* state, unsigned long seed, int size)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	if (id < size)
	{
		hiprand_init(seed, id, 0, &state[id]);
	}
}