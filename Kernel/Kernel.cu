#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""
#include "./Kernel.cuh"
#include "../Selections/RandomSelection.h"
#include "../Crossovers/IntervalCrossover.h"
#include "../Mutations/SwapMutation.h"

// Fisher-Yates shuffle algorithm
__device__ void shuffleChromosome(int* chromosome, int size, hiprandState* state) {
	for (int i = size - 1; i > 0; i--) {
		int j = hiprand(state) % (i + 1);

		// Swap chromosome[i] with chromosome[j]
		int temp = chromosome[i];
		chromosome[i] = chromosome[j];
		chromosome[j] = temp;
	}
}

__global__ void geneticAlgorithmKernel(int** population, float** distance_matrix, int size, hiprandState* globalState, int max_iterations) {
	__shared__ float fitness[1024];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int populationSize = blockDim.x * gridDim.x;
	// Local hiprand state
	hiprandState localState = globalState[id];

	// Initialize chromosome with a sequence from 0 to size - 1
	for (int i = 0; i < size; ++i) {
		population[id][i] = i;
	}

	shuffleChromosome(population[id], size, &localState);
	fitness[id] = calculateFitness(population[id], size, distance_matrix);
	__syncthreads();

	// Run the genetic algorithm for a fixed number of iterations
	for (int iteration = 0; iteration < max_iterations; ++iteration) {
		// Selection
		// Assuming a simplistic random selection for demonstration
		int otherIdToSelection = hiprand(&localState) % populationSize;
		selection(population[id], population[otherIdToSelection], size, fitness[id], fitness[otherIdToSelection]);

		__syncthreads(); // Synchronize after selection for crossover

		int otherIdToCrossover = hiprand(&localState) % populationSize;
		int* child = crossover(population[id], population[otherIdToCrossover], size, &localState);
		for (int i = 0; i < size; ++i) {
			population[id][i] = child[i];
		}
		delete[] child;

		// Mutation
		mutate(population[id * size], size, &localState);
		fitness[id] = calculateFitness(population[id], size, distance_matrix);
		__syncthreads(); // Synchronize after mutation
	}
	// Update the global state to ensure randomness continuity
	globalState[id] = localState;
}

__global__ void tspGeneticAlgorithmKernel(int** population, float** distance_matrix, int size, hiprandState* globalState, int max_iterations) {
	__shared__ float fitness[1024];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int populationSize = blockDim.x * gridDim.x;
	// Local hiprand state
	hiprandState localState = globalState[id];

	// Initialize chromosome with a sequence from 0 to size - 1
	for (int i = 0; i < size; ++i) {
		population[id][i] = i;
	}

	shuffleChromosome(population[id], size, &localState);
	fitness[id] = calculateFitness(population[id], size, distance_matrix);
	__syncthreads();

	for (int iteration = 0; iteration < max_iterations; ++iteration) {
		// Selection - using roulette wheel to select an index
		int selectedIdx = rouletteWheelSelection(fitness, populationSize, &localState);

		// Crossover - Order Crossover (OX)
		int* child = crossover(population[id], population[selectedIdx], size, &localState);
		for (int i = 0; i < size; ++i) {
			population[id][i] = child[i];
		}
		delete[] child;

		__syncthreads(); // Synchronize threads before mutation

		// Mutation - Inversion Mutation
		intervalMutate(population[id], size, &localState);

		// Calculate fitness of the new chromosome
		fitness[id] = calculateFitness(population[id], size, distance_matrix);

		__syncthreads(); // Synchronize threads after mutation
	}

	// Update the global state to ensure randomness continuity
	globalState[id] = localState;
}